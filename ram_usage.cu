#include <iostream>
#include <unistd.h>
#include "hip/hip_runtime.h"

int main()
{
    // show memory usage of GPU
    size_t free_byte ;
    size_t total_byte ;

    while (true )
    {
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

        if ( hipSuccess != cuda_status ){
            std::cout << "Error: hipMemGetInfo fails, "
                      << hipGetErrorString(cuda_status) << std::endl;
            exit(1);
        }

        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;

        std::cout << "GPU memory usage: used = " << used_db/1024.0/1024.0 << ", free = "
                  << free_db/1024.0/1024.0 << " MB, total = " << total_db/1024.0/1024.0 << " MB"
                  << std::endl; sleep(1);
        break;
    }

    return 0;
}
